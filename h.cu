
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>


using namespace std;

__global__ void add(int *a, int *b, int *c, int n){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < n){
		c[index] = a[index] + b[index];
	}
}

int main(int argc, char *argv[]){
	cout<<"****** Array Addition ******\n"<<endl;
	int *a, *b, *c;
	int size = 10;
	a = (int*) malloc(size*sizeof(int));
	b = (int*) malloc(size*sizeof(int));
	c = (int*) malloc(size*sizeof(int));

	int *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	

	add<<<1,10>>>(d_a, d_b, d_c, size);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(a);
	free(b);
	free(c);
	return 0;
}

